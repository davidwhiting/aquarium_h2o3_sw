#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>

#define N 16

// TODO make this method a kernel
void add(int* a, int* b, int* c) {
    int localIdx = // TODO calculate the index for this kernel invocation using read-only CUDA provided variables
    
    // TODO when should we do the summation? What values can localIdx take compared to N?
    if( ... ) {
        c[localIdx] = a[localIdx] + b[localIdx];
    }
}

int main( int argc, char** argv ) {
    int a[N], b[N], c[N]; 
    int *dev_a, *dev_b, *dev_c;

    // Initialize arrays a and b with data
    for (int i=0; i < N; i++) {
        a[i] = 2*i;
        b[i] = -i;
    }
    
    // Allocate memory on the GPU
    hipMalloc( (void**)&dev_a, N * sizeof(int) ); 
    hipMalloc( (void**)&dev_b, N * sizeof(int) );
    hipMalloc( (void**)&dev_c, N * sizeof(int) );
    
    // Copy the data from host to GPU memory
    hipMemcpy( dev_a, a, N * sizeof(int), hipMemcpyHostToDevice );
    hipMemcpy( dev_b, b, N * sizeof(int), hipMemcpyHostToDevice );

    // Compute the number of block necessary based on a constant number of threads per block
    // Be careful - this can launch more threads than we need, we need to handle this in the kernel!
    int threadsPerBlock = 1024;
    int blocks = (int)ceil((float)N/threadsPerBlock);

    // Launch the kernel
    add<<<blocks,threadsPerBlock>>>(dev_a, dev_b, dev_c);
    
    // Move the result back from the GPU to the host
    hipMemcpy( c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost );
    
    for (int i=0; i < N; i++) {
        printf( "%d + %d = %d\n", a[i], b[i], c[i] );
    }
 
    // Always free the memory you explicitly allocated
    hipFree( dev_a );
    hipFree( dev_b );
    hipFree( dev_c );

    return 0;
}
