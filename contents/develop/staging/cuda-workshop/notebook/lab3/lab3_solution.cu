
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

#define N 16

__global__ void add(int* a, int* b, int* c) {
    int localIdx = blockIdx.x*blockDim.x + threadIdx.x;
    
    if(localIdx < N) {
        c[localIdx] = a[localIdx] + b[localIdx];
    }
}

int main( int argc, char** argv ) {
    int a[N], b[N], c[N]; 
    int *dev_a, *dev_b, *dev_c;

    // Initialize arrays a and b with data
    for (int i=0; i < N; i++) {
        a[i] = 2*i;
        b[i] = -i;
    }
    
    // Allocate memory on the GPU
    hipMalloc( (void**)&dev_a, N * sizeof(int) ); 
    hipMalloc( (void**)&dev_b, N * sizeof(int) );
    hipMalloc( (void**)&dev_c, N * sizeof(int) );
    
    // Copy the data from host to GPU memory
    hipMemcpy( dev_a, a, N * sizeof(int), hipMemcpyHostToDevice );
    hipMemcpy( dev_b, b, N * sizeof(int), hipMemcpyHostToDevice );

    // Compute the number of block necessary based on a constant number of threads per block
    // Be careful - this can launch more threads than we need, we need to handle this in the kernel!
    int threadsPerBlock = 1024;
    int blocks = (int)ceil((float)N/threadsPerBlock);

    // Launch the kernel
    add<<<blocks,threadsPerBlock>>>(dev_a, dev_b, dev_c);

    hipError_t syncErrCode = hipGetLastError();
    hipError_t asyncErrCode = hipDeviceSynchronize();
    if (syncErrCode != hipSuccess) { printf("Kernel error: %s\n", hipGetErrorString(syncErrCode)); }
    if (asyncErrCode != hipSuccess) { printf("Kernel error: %s\n", hipGetErrorString(asyncErrCode)); }

    // Move the result back from the GPU to the host
    hipMemcpy( c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost );
    
    for (int i=0; i < N; i++) {
        printf( "%d + %d = %d\n", a[i], b[i], c[i] );
    }
 
    // Always free the memory you explicitly allocated
    hipFree( dev_a );
    hipFree( dev_b );
    hipFree( dev_c );

    return 0;
}

